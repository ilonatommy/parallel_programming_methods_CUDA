#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "hip/hip_runtime_api.h"
#include "helper_timer.h"

__global__ void d_add (int *a, int *b, int *c, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < N) {
        c[tid] = a[tid]+b[tid];
    }
}

void h_add(int *a, int *b, int *c, int N)
{
    int i;
    for(i=0; i<N; i++)
    {
        c[i] = a[i] + b[i];
    }
}

void d_allocate_vector(int **v, int length)
{
    hipError_t err = hipSuccess;
    err = hipMalloc((void**)v,length * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void h_verify_equal(int* v1, int* v2, int length)
{
    for (int i = 0; i < length; i++)
    {
    	if (fabs(v1[i] - v2[i] > 1e-5))
    	{
    		fprintf(stderr, "Result verification failed at element %d!\n", i);
        	exit(EXIT_FAILURE);
    	}
	}
    printf("Test PASSED\n");
}

void h_check_kernel_errors()
{
    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void h_copy2host(int* from, int* to, int length)
{
    hipError_t err = hipSuccess;
    err = hipMemcpy(to, from, length*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char *argv[]) {
    int N = atoi(argv[1]);
    
    int a[N], b[N], c[N], ctrl_c[N];
    int *dev_a, *dev_b, *dev_c;
    
    //allocate memory - I added allocation success check
    d_allocate_vector(&dev_a, N);
    d_allocate_vector(&dev_b, N);
    d_allocate_vector(&dev_c, N);
    
    // initialise variables on the host
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i*2;
    }
    
    // copy the host input vectors in host memory to the device input vectors in device memory
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);
    
    //timer start:
    StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);
    
    // launch kernel, check if succeded
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;;
    d_add <<<blocksPerGrid,threadsPerBlock>>> (dev_a,dev_b,dev_c, N);
    h_check_kernel_errors();
    
    //timer stop:
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    float d_time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    
    // the same code in host version:
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);    
    h_add(a, b, ctrl_c, N);    
    float h_time = sdkGetTimerValue(&timer);
    
    // Copy the device result vector in device memory to the host result vector in host memory.
    h_copy2host(c, dev_c, N);
        
    // Verify that the result vector is correct
    h_verify_equal(ctrl_c, c, N);
    
    if(N<=10)
    {
        for (int i = 0; i < N; i++) printf("%d+%d=%d\n", a[i], b[i], c[i]);
    }
    
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // print results:
    printf ("Time for the device: %f ms, fot the host: %f ms.\n", d_time, h_time); 
    
    return 0;
}
