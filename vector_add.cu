 
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#define N 10

__global__ void add (int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < N) {
        c[tid] = a[tid]+b[tid];
    }
}

int main(void) {
    hipError_t err = hipSuccess;
    
    int a[N],b[N],c[N];
    int *dev_a, *dev_b, *dev_c;
    
    //allocate memory - I added allocation success check
    err = hipMalloc((void**)&dev_a,N * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void**)&dev_b,N * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void**)&dev_c,N * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // initialise variables on the host
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i*2;
    }
    
    // copy the host input vectors in host memory to the device input vectors in device memory
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);
    
    // launch kernel, check if succeded
    int blocksPerGrid = 1;
    int threadsPerBlock = N;
    add <<<blocksPerGrid,threadsPerBlock>>> (dev_a,dev_b,dev_c);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Copy the device result vector in device memory to the host result vector in host memory.
    err = hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    for (int i = 0; i < N; i++) {
        printf("%d+%d=%d\n", a[i], b[i], c[i]);
    }
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    return 0;
}
